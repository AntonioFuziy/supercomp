#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
 #include <thrust/host_vector.h>
 #include <thrust/device_vector.h>
 #include <thrust/generate.h>
 #include <thrust/functional.h>
 #include <thrust/copy.h>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <cmath>
#include <random>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
  auto ms = duration_cast<milliseconds>(span);
  std::cout << msg << " - levou - " <<
  ms.count() << " milisegundos" << std::endl;
}

struct square
{
  __host__ __device__
  float operator()(const int& x) {
    return x*x;
  }
};

// CRIE UMA FUNCTOR PARA CALCULAR A SQUARE

 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<int> a, thrust::device_vector<int> b) {
  float result;
  // ... add Thrust calls
  // AQUI VAO AS CHAMADAS THRUST 
  thrust::transform(
    a.begin(), a.end(), 
    b.begin(), b.end(),
    square(a)
  );

  result = std::sqrt(thrust::reduce(b.begin(), b.end(), (float) 0, thrust::plus<float>()));

  return result;
};

int main(int argc, char** argv) {
  if (argc != 2) {
    std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
    std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
    return 1;
  }
  int n = std::atoi(argv[1]); //numero de elementos
  steady_clock::time_point ts, te;

  // Faça um  vector em thrust 

  thrust::device_vector<int> d_a(n);
  thrust::device_vector<int> d_b(n);

  // inicilize o  vector
  ts = steady_clock::now();

  thrust::generate(d_a.begin(), d_a.end(), std::rand);

  te = steady_clock::now();
  reportTime("Inicializacao", te - ts);

  // Calcule a magnitude do vetor
  ts = steady_clock::now();
  float len = magnitude(d_a, d_b);
  te = steady_clock::now();
  reportTime("Tempo para calculo", te - ts);


  std::cout << std::fixed << std::setprecision(4);
  std::cout << "Magnitude : " << len << std::endl;
}
