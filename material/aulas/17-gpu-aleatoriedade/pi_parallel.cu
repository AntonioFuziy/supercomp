#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <vector>

struct rng_gpu {  
    __device__ __host__
    double operator() (const int n) {
        thrust::default_random_engine eng(n);
        thrust::uniform_real_distribution<double> d(25, 40);
        return d(eng);
    }
};

int main(int argc, char* argv[]) {
    size_t N = 10000;
    thrust::host_vector<double> cpu(2*N);
    thrust::device_vector<double> gpu = cpu;

    thrust::transform(
        thrust::make_counting_iterator<int>(0),
        thrust::make_counting_iterator<int>(2*N),
        gpu.begin(),
        rng_gpu()
    );

    int sum = 0;
    double dist = 0;
    for(int i = 0; i < N; i++){
        dist = pow(gpu[i],2) + pow(gpu[2*i],2);
        if(dist <= 1){
            sum += 1;
        }
    }

    double pi = 4*sum/N;
    std::cout << "pi: " << pi << std::endl;

    return 0;
}