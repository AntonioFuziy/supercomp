
// Exemplo de calculo de normal

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

int main (void){
    hipblasHandle_t handle;
    float *devPtrA;
    
    float a[3] = {1,1,1};
    float b;
    
    hipMalloc((void**)&devPtrA, 3*sizeof(float));
    
    hipblasCreate(&handle);
    
    hipblasSetVector(3, sizeof(float), &a, 1, devPtrA, 1);
    
    hipblasSnrm2(handle, 3, devPtrA, 1, &b);
    
    hipFree(devPtrA);
    hipblasDestroy(handle);

    printf ("%f", b);
    
    return EXIT_SUCCESS;
}