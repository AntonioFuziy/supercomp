
// Exemplo de produto escalar

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

int main (void){
    hipblasHandle_t handle;

    float *devPtrA;
    float *devPtrB;

    float a[3] = {2,0,0};
    float b[3] = {3,0,0};        
    float c;

    hipMalloc((void**)&devPtrA, 3*sizeof(float));
    hipMalloc((void**)&devPtrB, 3*sizeof(float));
    
    hipblasCreate(&handle);
    
    hipblasSetVector(3, sizeof(float), &a, 1, devPtrA, 1);
    hipblasSetVector(3, sizeof(float), &b, 1, devPtrB, 1);
    
    hipblasSdot(handle, 3, devPtrA, 1, devPtrB, 1, &c);
        
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipblasDestroy(handle);
    
    printf ("%3.0f", c);

    return EXIT_SUCCESS;
}