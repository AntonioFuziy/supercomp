
// Exemplo de operações em vetores

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 5

int main (void){
    hipblasHandle_t handle;
    float* devPtrA;
    float* a = 0;
    int i;

    a = (float *)malloc (M * sizeof (float));
   
    for (i = 0; i < M; i++)
            a[i] = (float)rand();

    hipMalloc((void**)&devPtrA, M*sizeof(float));
    
    hipblasCreate(&handle);
    
    hipblasSetVector(M, sizeof(float), a, 1, devPtrA, 1);

    int maximo;
    hipblasIsamax(handle, M, devPtrA, 1, &maximo);
    printf("max %d\n",maximo);

    int minimo;
    hipblasIsamin(handle, M, devPtrA, 1, &minimo);
    printf("min %d\n",minimo);
    
    float soma;
    hipblasSasum(handle, M, devPtrA, 1, &soma);
    printf("soma %5.0f\n",soma);

    hipFree (devPtrA);
    hipblasDestroy(handle);

    for (i = 0; i < M; i++)
        printf ("%5.0f  ", a[i]);

    free(a);
    return EXIT_SUCCESS;
}